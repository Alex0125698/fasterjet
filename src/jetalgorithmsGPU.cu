#include "hip/hip_runtime.h"
/**
 * \file jetalgorithmsGPU.cu (description...)
 * 
 * \author A.S. Woodcock
 * 
 * \licence GPL3 (see COPYING.md)
*/

#include "fasterjet/precompiledheader.hpp"
#include "fasterjet/jetalgorithmsGPU.hpp"
#include "fasterjet/jetalgorithmsCPU.hpp"
#include "fasterjet/utils.hpp"
#include "fasterjet/jetPrivate.hpp"
#include "fasterjet/utils_cuda.hpp"

// cuda block size
constexpr int BLOCK_SIZE = 256;

// generate N particles with randomized states
void makeRandomParticlesGPU(JetDataGPU& d, const int N)
{
   vector<Particle> particles(N);
   for (auto& p : particles)
   {
      p.pt = random(1,10);
      p.eta = random(-6,+6);
      p.phi = random(0,2*PI);
   }
   d.p->init(particles);
}

// square of x
__device__ inline float sqr(const float x)
{
   return x*x;
}

// extend JetDataPrivate with GPU-specific stuff

struct JetDataPrivateGPU : public JetDataPrivate
{ 
   void free()
   {
      JetDataPrivate::free();
      if (gpu_particles != nullptr) hipFree(gpu_particles);
      if (gpu_mindists != nullptr) hipFree(gpu_mindists);
      if (gpu_others != nullptr) hipFree(gpu_others);
      gpu_particles = nullptr;
      gpu_mindists = nullptr;
      gpu_others = nullptr;
   }

   void init(const std::vector<Particle>& p)
   {
      JetDataPrivate::init(p);

      if (pnReserve < nReserve)
      {
         // free();
         assert (gpu_particles == nullptr);
         assert (gpu_mindists == nullptr);
         assert (gpu_others == nullptr);
         hipMalloc(&gpu_particles, nReserve*sizeof(Particle));
         hipMalloc(&gpu_mindists, nReserve*sizeof(float));
         hipMalloc(&gpu_others, nReserve*sizeof(int));
         pnReserve = nReserve;
      }
   }

   // fill in gpu_particles from particles
   void copyCPU2GPU()
   {
      hipMemcpy(gpu_particles, &(particles[0]), nParticles * sizeof(Particle), hipMemcpyHostToDevice);
   }

   // fill in mindists, others from gpu_mindists, gpu_others
   void copyGPU2CPU()
   {
      hipMemcpy(&(mindists[0]), gpu_mindists, nParticles * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(&(others[0]), gpu_others, nParticles * sizeof(int), hipMemcpyDeviceToHost);
   }

};

JetDataGPU::JetDataGPU()
{
   p = new JetDataPrivateGPU;
}

// uses asymmetrical distance and separate Particle struct
__global__ void closest_finder_1(const int nParticles, const float R, const Particle* __restrict const particles, 
                                 float* __restrict const mindists, int* __restrict const others)
{
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index >= nParticles) return; 
   
   size_t i = index;

   // initialize with beam axis distance
   mindists[i] = 1./sqr(particles[index].pt);
   others[i] = -1;

   // check all distances to tother particles and get minimum
   // no improvement from this...
   float mindist = mindists[i];
   int other = others[i];
   auto pi = particles[i];

   for (size_t j=0; j<nParticles; ++j)
   {
      auto pj = particles[j];

      float delta_phi = abs(pi.phi - pj.phi); 
      delta_phi = min(delta_phi, 2*PI-delta_phi);
      float sqRij = sqr(pi.eta - pj.eta) + sqr(delta_phi);
      float dist = (sqRij / sqr(R)) / sqr(pi.pt);

      // some improvement...
      bool shouldSwap = dist < mindist && i != j;
      mindist = shouldSwap ? dist : mindist;
      other   = shouldSwap ?    j : other;
   }

   mindists[i] = mindist;
   others[i] = other;
}

// ...
void closest_finder_wrapper(JetDataPrivateGPU& d)
{
      // copy particle array to device
      d.copyCPU2GPU();

      // launch the kernel
      const int N_BLOCKS = 1+(d.nIniParticles + BLOCK_SIZE - 1) / BLOCK_SIZE;
      closest_finder_1<<<N_BLOCKS,BLOCK_SIZE>>>(d.nParticles, d.R, d.gpu_particles, d.gpu_mindists, d.gpu_others);

      // Wait for GPU to finish before accessing on host
      // apparently we dont need it here
      // hipDeviceSynchronize();

      // copy results to host
      d.copyGPU2CPU();
}


void findJetsGPU1(JetDataGPU& d, const float R)
{
   jet_recombiner_CPU(*d.p,R,(closest_finder_alg)closest_finder_wrapper);
}
